from google.colab import files

# Upload the image
uploaded = files.upload()

# Get the file name
image_filename = list(uploaded.keys())[0]

%%writefile cuda_code.cu

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

_global_ void flip_horizontal_chunk(int *img_chunk, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int row_start = i * cols;
    int row_end = (i + 1) * cols - 1;
    while (row_start < row_end) {
        int temp = img_chunk[row_start];
        img_chunk[row_start] = img_chunk[row_end];
        img_chunk[row_end] = temp;
        row_start++;
        row_end--;
    }
}

void load_image(int *image_data, int rows, int cols, const char *file_path) {
    FILE *file = fopen(file_path, "rb");
    if (!file) {
        perror("Error opening file");
        exit(EXIT_FAILURE);
    }

    fread(image_data, sizeof(int), rows * cols, file);

    fclose(file);
}

int main() {
    int rows, cols;
    int *image_data = NULL;

    const char *uploaded_file_path = "me.jpg";  // Set the uploaded file path
    rows = 1000;  // Adjust based on your image dimensions
    cols = 1000;  // Adjust based on your image dimensions
    size_t img_size = rows * cols * sizeof(int);
    image_data = (int *)malloc(img_size);
    load_image(image_data, rows, cols, uploaded_file_path);  // Use the uploaded image path

    // Sequential Part
    hipEvent_t startSeq, stopSeq;
    hipEventCreate(&startSeq);
    hipEventCreate(&stopSeq);
    hipEventRecord(startSeq);

    // ... Your sequential code here ...

    hipEventRecord(stopSeq);
    hipEventSynchronize(stopSeq);
    float seqElapsedTime;
    hipEventElapsedTime(&seqElapsedTime, startSeq, stopSeq);

    printf("Sequential Time: %f ms\n", seqElapsedTime);

    // Parallel Part
    int *d_image_data;
    hipMalloc((void **)&d_image_data, img_size);

    hipMemcpy(d_image_data, image_data, img_size, hipMemcpyHostToDevice);

    hipEvent_t startPar, stopPar;
    hipEventCreate(&startPar);
    hipEventCreate(&stopPar);
    hipEventRecord(startPar);

    int block_size = 256;
    int grid_size = (rows + block_size - 1) / block_size;

    flip_horizontal_chunk<<<grid_size, block_size>>>(d_image_data, rows, cols);

    hipEventRecord(stopPar);
    hipEventSynchronize(stopPar);
    float parElapsedTime;
    hipEventElapsedTime(&parElapsedTime, startPar, stopPar);

    printf("Parallel Time: %f ms\n", parElapsedTime);

    hipMemcpy(image_data, d_image_data, img_size, hipMemcpyDeviceToHost);

    hipFree(d_image_data);

    free(image_data);

    return 0;
}
